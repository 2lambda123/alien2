#include "hip/hip_runtime.h"
#include "CudaSimulationFacade.cuh"

#include <functional>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <>
#include <cuda/hip/hip_runtime_api.h>

#include "Base/Exceptions.h"
#include "Base/LoggingService.h"

#include "EngineInterface/InspectedEntityIds.h"
#include "EngineInterface/SimulationParameters.h"
#include "EngineInterface/GpuSettings.h"
#include "EngineInterface/SpaceCalculator.h"

#include "DataAccessKernels.cuh"
#include "TOs.cuh"
#include "Base.cuh"
#include "GarbageCollectorKernels.cuh"
#include "ConstantMemory.cuh"
#include "CudaMemoryManager.cuh"
#include "SimulationStatistics.cuh"
#include "Objects.cuh"
#include "Map.cuh"
#include "StatisticsKernels.cuh"
#include "EditKernels.cuh"
#include "RenderingKernels.cuh"
#include "SimulationData.cuh"
#include "SimulationKernelsLauncher.cuh"
#include "DataAccessKernelsLauncher.cuh"
#include "RenderingKernelsLauncher.cuh"
#include "EditKernelsLauncher.cuh"
#include "StatisticsKernelsLauncher.cuh"
#include "SelectionResult.cuh"
#include "RenderingData.cuh"
#include "TestKernelsLauncher.cuh"

namespace
{
    class CudaInitializer
    {
    public:
        static void init() { getInstance(); }
        static std::string getGpuName() { return getInstance()._gpuName; }

        CudaInitializer()
        {
            int deviceNumber = getDeviceNumberOfHighestComputeCapability();

            auto result = hipSetDevice(deviceNumber);
            if (result != hipSuccess) {
                throw SystemRequirementNotMetException("CUDA device could not be initialized.");
            }

            std::stringstream stream;
            stream << "device " << deviceNumber << " is set";
            log(Priority::Important, stream.str());
        }

        ~CudaInitializer() { hipDeviceReset(); }

    private:
        static CudaInitializer& getInstance()
        {
            static CudaInitializer instance;
            return instance;
        }

        int getDeviceNumberOfHighestComputeCapability()
        {
            int result = 0;
            int numberOfDevices;
            CHECK_FOR_CUDA_ERROR(hipGetDeviceCount(&numberOfDevices));
            if (numberOfDevices < 1) {
                throw SystemRequirementNotMetException("No CUDA device found.");
            }
            {
                std::stringstream stream;
                if (1 == numberOfDevices) {
                    stream << "1 CUDA device found";
                } else {
                    stream << numberOfDevices << " CUDA devices found";
                }
                log(Priority::Important, stream.str());
            }

            int highestComputeCapability = 0;
            for (int deviceNumber = 0; deviceNumber < numberOfDevices; ++deviceNumber) {
                hipDeviceProp_t prop;
                CHECK_FOR_CUDA_ERROR(hipGetDeviceProperties(&prop, deviceNumber));

                std::stringstream stream;
                stream << "device " << deviceNumber << ": " << prop.name << " with compute capability " << prop.major
                       << "." << prop.minor;
                log(Priority::Important, stream.str());

                int computeCapability = prop.major * 100 + prop.minor;
                if (computeCapability > highestComputeCapability) {
                    result = deviceNumber;
                    highestComputeCapability = computeCapability;
                    _gpuName = prop.name;
                }
            }
            if (highestComputeCapability < 600) {
                throw SystemRequirementNotMetException(
                    "No CUDA device with compute capability of 6.0 or higher found.");
            }

            return result;
        }

        std::string _gpuName;
    };
}

void _CudaSimulationFacade::initCuda()
{
    CudaInitializer::init();
}

_CudaSimulationFacade::_CudaSimulationFacade(uint64_t timestep, Settings const& settings)
{
    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    _settings.generalSettings = settings.generalSettings;
    setSimulationParameters(settings.simulationParameters);
    setGpuConstants(settings.gpuSettings);

    log(Priority::Important, "initialize simulation");

    _cudaSimulationData = std::make_shared<SimulationData>();
    _cudaRenderingData = std::make_shared<RenderingData>();
    _cudaSelectionResult = std::make_shared<SelectionResult>();
    _cudaAccessTO = std::make_shared<DataTO>();
    _simulationStatistics = std::make_shared<SimulationStatistics>();

    _cudaSimulationData->init({settings.generalSettings.worldSizeX, settings.generalSettings.worldSizeY}, timestep);
    _cudaRenderingData->init();
    _simulationStatistics->init();
    _cudaSelectionResult->init();

    _simulationKernels = std::make_shared<_SimulationKernelsLauncher>();
    _dataAccessKernels = std::make_shared<_DataAccessKernelsLauncher>();
    _garbageCollectorKernels = std::make_shared<_GarbageCollectorKernelsLauncher>();
    _renderingKernels = std::make_shared<_RenderingKernelsLauncher>();
    _editKernels = std::make_shared<_EditKernelsLauncher>();
    _statisticsKernels = std::make_shared<_StatisticsKernelsLauncher>();

    CudaMemoryManager::getInstance().acquireMemory<uint64_t>(1, _cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().acquireMemory<uint64_t>(1, _cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().acquireMemory<uint64_t>(1, _cudaAccessTO->numAuxiliaryData);

    //default array sizes for empty simulation (will be resized later if not sufficient)
    resizeArrays({100000, 100000, 100000});
}

_CudaSimulationFacade::~_CudaSimulationFacade()
{
    _cudaSimulationData->free();
    _cudaRenderingData->free();
    _simulationStatistics->free();
    _cudaSelectionResult->free();

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->auxiliaryData);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numAuxiliaryData);

    log(Priority::Important, "close simulation");
}

void* _CudaSimulationFacade::registerImageResource(GLuint image)
{
    hipGraphicsResource* cudaResource;

    CHECK_FOR_CUDA_ERROR(
        hipGraphicsGLRegisterImage(&cudaResource, image, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly));

    return reinterpret_cast<void*>(cudaResource);
}

std::string _CudaSimulationFacade::getGpuName()
{
    return CudaInitializer::getGpuName();
}

void _CudaSimulationFacade::calcTimestep()
{
    checkAndProcessSimulationParameterChanges();

    Settings settings = [this] {
        std::lock_guard lock(_mutexForSimulationParameters);
        _simulationKernels->calcSimulationParametersForNextTimestep(_settings);
        CHECK_FOR_CUDA_ERROR(
            hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &_settings.simulationParameters, sizeof(SimulationParameters), 0, hipMemcpyHostToDevice));
        return _settings;
    }();

    _simulationKernels->calcTimestep(settings, getSimulationDataIntern(), *_simulationStatistics);
    syncAndCheck();

    automaticResizeArrays();

    std::lock_guard lock(_mutexForSimulationData);
    ++_cudaSimulationData->timestep;
}

void _CudaSimulationFacade::applyCataclysm(int power)
{
    for (int i = 0; i < power; ++i) {
        _editKernels->applyCataclysm(_settings.gpuSettings, getSimulationDataIntern());
        syncAndCheck();
        resizeArraysIfNecessary();
    }
}

void _CudaSimulationFacade::drawVectorGraphics(
    float2 const& rectUpperLeft,
    float2 const& rectLowerRight,
    void* cudaResource,
    int2 const& imageSize,
    double zoom)
{
    checkAndProcessSimulationParameterChanges();

    auto cudaResourceImpl = reinterpret_cast<hipGraphicsResource*>(cudaResource);
    CHECK_FOR_CUDA_ERROR(hipGraphicsMapResources(1, &cudaResourceImpl));

    hipArray* mappedArray;
    CHECK_FOR_CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&mappedArray, cudaResourceImpl, 0, 0));

    _cudaRenderingData->resizeImageIfNecessary(imageSize);

    _renderingKernels->drawImage(
        _settings.gpuSettings, rectUpperLeft, rectLowerRight, imageSize, static_cast<float>(zoom), getSimulationDataIntern(), *_cudaRenderingData);
    syncAndCheck();

    const size_t widthBytes = sizeof(uint64_t) * imageSize.x;
    CHECK_FOR_CUDA_ERROR(hipMemcpy2DToArray(
        mappedArray,
        0,
        0,
        _cudaRenderingData->imageData,
        widthBytes,
        widthBytes,
        imageSize.y,
        hipMemcpyDeviceToDevice));

    CHECK_FOR_CUDA_ERROR(hipGraphicsUnmapResources(1, &cudaResourceImpl));
}

void _CudaSimulationFacade::getSimulationData(
    int2 const& rectUpperLeft,
    int2 const& rectLowerRight,
    DataTO const& dataTO)
{
    _dataAccessKernels->getData(_settings.gpuSettings, getSimulationDataIntern(), rectUpperLeft, rectLowerRight, *_cudaAccessTO);
    syncAndCheck();

    copyDataTOtoHost(dataTO);
}

void _CudaSimulationFacade::getSelectedSimulationData(bool includeClusters, DataTO const& dataTO)
{
    _dataAccessKernels->getSelectedData(_settings.gpuSettings, getSimulationDataIntern(), includeClusters, *_cudaAccessTO);
    syncAndCheck();

    copyDataTOtoHost(dataTO);
}

void _CudaSimulationFacade::getInspectedSimulationData(std::vector<uint64_t> entityIds, DataTO const& dataTO)
{
    InspectedEntityIds ids;
    if (entityIds.size() > Const::MaxInspectedObjects) {
        return;
    }
    for (int i = 0; i < entityIds.size(); ++i) {
        ids.values[i] = entityIds.at(i);
    }
    if (entityIds.size() < Const::MaxInspectedObjects) {
        ids.values[entityIds.size()] = 0;
    }
    _dataAccessKernels->getInspectedData(_settings.gpuSettings, getSimulationDataIntern(), ids, *_cudaAccessTO);
    syncAndCheck();
    copyDataTOtoHost(dataTO);
}

void _CudaSimulationFacade::getOverlayData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataTO const& dataTO)
{
    _dataAccessKernels->getOverlayData(_settings.gpuSettings, getSimulationDataIntern(), rectUpperLeft, rectLowerRight, *_cudaAccessTO);
    syncAndCheck();

    copyToHost(dataTO.numCells, _cudaAccessTO->numCells);
    copyToHost(dataTO.numParticles, _cudaAccessTO->numParticles);
    copyToHost(dataTO.cells, _cudaAccessTO->cells, *dataTO.numCells);
    copyToHost(dataTO.particles, _cudaAccessTO->particles, *dataTO.numParticles);
}

void _CudaSimulationFacade::addAndSelectSimulationData(DataTO const& dataTO)
{
    copyDataTOtoDevice(dataTO);
    _editKernels->removeSelection(_settings.gpuSettings, getSimulationDataIntern());
    _dataAccessKernels->addData(_settings.gpuSettings, getSimulationDataIntern(), *_cudaAccessTO, true, true);
    syncAndCheck();
}

void _CudaSimulationFacade::setSimulationData(DataTO const& dataTO)
{
    copyDataTOtoDevice(dataTO);
    _dataAccessKernels->clearData(_settings.gpuSettings, getSimulationDataIntern());
    _dataAccessKernels->addData(_settings.gpuSettings, getSimulationDataIntern(), *_cudaAccessTO, false, false);
    syncAndCheck();
}

void _CudaSimulationFacade::removeSelectedObjects(bool includeClusters)
{
    _editKernels->removeSelectedObjects(_settings.gpuSettings, getSimulationDataIntern(), includeClusters);
    syncAndCheck();
}

void _CudaSimulationFacade::relaxSelectedObjects(bool includeClusters)
{
    _editKernels->relaxSelectedObjects(_settings.gpuSettings, getSimulationDataIntern(), includeClusters);
    syncAndCheck();
}

void _CudaSimulationFacade::uniformVelocitiesForSelectedObjects(bool includeClusters)
{
    _editKernels->uniformVelocities(_settings.gpuSettings, getSimulationDataIntern(), includeClusters);
    syncAndCheck();
}

void _CudaSimulationFacade::makeSticky(bool includeClusters)
{
    _editKernels->makeSticky(_settings.gpuSettings, getSimulationDataIntern(), includeClusters);
    syncAndCheck();
}

void _CudaSimulationFacade::removeStickiness(bool includeClusters)
{
    _editKernels->removeStickiness(_settings.gpuSettings, getSimulationDataIntern(), includeClusters);
    syncAndCheck();
}

void _CudaSimulationFacade::setBarrier(bool value, bool includeClusters)
{
    _editKernels->setBarrier(_settings.gpuSettings, getSimulationDataIntern(), value, includeClusters);
    syncAndCheck();
}

void _CudaSimulationFacade::changeInspectedSimulationData(DataTO const& changeDataTO)
{
    copyDataTOtoDevice(changeDataTO);
    _editKernels->changeSimulationData(_settings.gpuSettings, getSimulationDataIntern(), *_cudaAccessTO);
    syncAndCheck();

    resizeArraysIfNecessary();
}

void _CudaSimulationFacade::applyForce(ApplyForceData const& applyData)
{
    _editKernels->applyForce(_settings.gpuSettings, getSimulationDataIntern(), applyData);
    syncAndCheck();
}

void _CudaSimulationFacade::switchSelection(PointSelectionData const& pointData)
{
    _editKernels->switchSelection(_settings.gpuSettings, getSimulationDataIntern(), pointData);
    syncAndCheck();
}

void _CudaSimulationFacade::swapSelection(PointSelectionData const& pointData)
{
    _editKernels->swapSelection(_settings.gpuSettings, getSimulationDataIntern(), pointData);
    syncAndCheck();
}

void _CudaSimulationFacade::setSelection(AreaSelectionData const& selectionData)
{
    _editKernels->setSelection(_settings.gpuSettings, getSimulationDataIntern(), selectionData);
}

 SelectionShallowData _CudaSimulationFacade::getSelectionShallowData()
{
    _editKernels->getSelectionShallowData(_settings.gpuSettings, getSimulationDataIntern(), *_cudaSelectionResult);
    syncAndCheck();
    return _cudaSelectionResult->getSelectionShallowData();
}

void _CudaSimulationFacade::shallowUpdateSelectedObjects(ShallowUpdateSelectionData const& shallowUpdateData)
{
    _editKernels->shallowUpdateSelectedObjects(_settings.gpuSettings, getSimulationDataIntern(), shallowUpdateData);
    syncAndCheck();
}

void _CudaSimulationFacade::removeSelection()
{
    _editKernels->removeSelection(_settings.gpuSettings, getSimulationDataIntern());
    syncAndCheck();
}

void _CudaSimulationFacade::updateSelection()
{
    _editKernels->updateSelection(_settings.gpuSettings, getSimulationDataIntern());
    syncAndCheck();
}

void _CudaSimulationFacade::colorSelectedObjects(unsigned char color, bool includeClusters)
{
    _editKernels->colorSelectedCells(_settings.gpuSettings, getSimulationDataIntern(), color, includeClusters);
    syncAndCheck();
}

void _CudaSimulationFacade::reconnectSelectedObjects()
{
    _editKernels->reconnect(_settings.gpuSettings, getSimulationDataIntern());
    syncAndCheck();
}

void _CudaSimulationFacade::setDetached(bool value)
{
    _editKernels->setDetached(_settings.gpuSettings, getSimulationDataIntern(), value);
    syncAndCheck();
}

void _CudaSimulationFacade::setGpuConstants(GpuSettings const& gpuConstants)
{
    _settings.gpuSettings = gpuConstants;

    CHECK_FOR_CUDA_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(cudaThreadSettings), &gpuConstants, sizeof(GpuSettings), 0, hipMemcpyHostToDevice));
}

SimulationParameters _CudaSimulationFacade::getSimulationParameters() const
{
    std::lock_guard lock(_mutexForSimulationParameters);
    return _newSimulationParameters ? *_newSimulationParameters : _settings.simulationParameters;
}

void _CudaSimulationFacade::setSimulationParameters(SimulationParameters const& parameters)
{
    std::lock_guard lock(_mutexForSimulationParameters);
    _newSimulationParameters = parameters;
}

auto _CudaSimulationFacade::getArraySizes() const -> ArraySizes
{
    return {
        _cudaSimulationData->objects.cells.getSize_host(),
        _cudaSimulationData->objects.particles.getSize_host(),
        _cudaSimulationData->objects.auxiliaryData.getSize_host()
    };
}

StatisticsData _CudaSimulationFacade::getStatistics()
{
    _statisticsKernels->updateStatistics(_settings.gpuSettings, getSimulationDataIntern(), *_simulationStatistics);
    syncAndCheck();
    
    return _simulationStatistics->getStatistics();
}

void _CudaSimulationFacade::resetTimeIntervalStatistics()
{
    _simulationStatistics->resetAccumulatedStatistics();
}

uint64_t _CudaSimulationFacade::getCurrentTimestep() const
{
    std::lock_guard lock(_mutexForSimulationData);
    return _cudaSimulationData->timestep;
}

void _CudaSimulationFacade::setCurrentTimestep(uint64_t timestep)
{
    std::lock_guard lock(_mutexForSimulationData);
    _cudaSimulationData->timestep = timestep;
}

void _CudaSimulationFacade::clear()
{
    _dataAccessKernels->clearData(_settings.gpuSettings, getSimulationDataIntern());
    syncAndCheck();
}

void _CudaSimulationFacade::resizeArraysIfNecessary(ArraySizes const& additionals)
{
    if (_cudaSimulationData->shouldResize(additionals)) {
        resizeArrays(additionals);
    }
}

void _CudaSimulationFacade::testOnly_mutate(uint64_t cellId, MutationType mutationType)
{
    _testKernels->testOnly_mutate(_settings.gpuSettings, getSimulationDataIntern(), cellId, mutationType);
    syncAndCheck();

    resizeArraysIfNecessary();
}

void _CudaSimulationFacade::syncAndCheck()
{
    hipDeviceSynchronize();
    CHECK_FOR_CUDA_ERROR(hipGetLastError());
}

void _CudaSimulationFacade::copyDataTOtoDevice(DataTO const& dataTO)
{
    copyToDevice(_cudaAccessTO->numCells, dataTO.numCells);
    copyToDevice(_cudaAccessTO->numParticles, dataTO.numParticles);
    copyToDevice(_cudaAccessTO->numAuxiliaryData, dataTO.numAuxiliaryData);

    copyToDevice(_cudaAccessTO->cells, dataTO.cells, *dataTO.numCells);
    copyToDevice(_cudaAccessTO->particles, dataTO.particles, *dataTO.numParticles);
    copyToDevice(_cudaAccessTO->auxiliaryData, dataTO.auxiliaryData, *dataTO.numAuxiliaryData);
}

void _CudaSimulationFacade::copyDataTOtoHost(DataTO const& dataTO)
{
    copyToHost(dataTO.numCells, _cudaAccessTO->numCells);
    copyToHost(dataTO.numParticles, _cudaAccessTO->numParticles);
    copyToHost(dataTO.numAuxiliaryData, _cudaAccessTO->numAuxiliaryData);

    copyToHost(dataTO.cells, _cudaAccessTO->cells, *dataTO.numCells);
    copyToHost(dataTO.particles, _cudaAccessTO->particles, *dataTO.numParticles);
    copyToHost(dataTO.auxiliaryData, _cudaAccessTO->auxiliaryData, *dataTO.numAuxiliaryData);
}

void _CudaSimulationFacade::automaticResizeArrays()
{
    uint64_t timestep;
    {
        std::lock_guard lock(_mutexForSimulationData);
        timestep = _cudaSimulationData->timestep;
    }
    //make check after every 10th time step
    if (timestep % 10 == 0) {
        resizeArraysIfNecessary();
    }
}

void _CudaSimulationFacade::resizeArrays(ArraySizes const& additionals)
{
    log(Priority::Important, "resize arrays");

    _cudaSimulationData->resizeTargetObjects(additionals);
    if (!_cudaSimulationData->isEmpty()) {
        _garbageCollectorKernels->copyArrays(_settings.gpuSettings, getSimulationDataIntern());
        syncAndCheck();

        _cudaSimulationData->resizeObjects();

        _garbageCollectorKernels->swapArrays(_settings.gpuSettings, getSimulationDataIntern());
        syncAndCheck();
    } else {
        _cudaSimulationData->resizeObjects();
    }

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->auxiliaryData);

    auto cellArraySize = _cudaSimulationData->objects.cells.getSize_host();
    CudaMemoryManager::getInstance().acquireMemory<CellTO>(cellArraySize, _cudaAccessTO->cells);
    auto particleArraySize = _cudaSimulationData->objects.particles.getSize_host();
    CudaMemoryManager::getInstance().acquireMemory<ParticleTO>(particleArraySize, _cudaAccessTO->particles);
    auto auxiliaryDataSize = _cudaSimulationData->objects.auxiliaryData.getSize_host();
    CudaMemoryManager::getInstance().acquireMemory<uint8_t>(auxiliaryDataSize, _cudaAccessTO->auxiliaryData);

    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    log(Priority::Unimportant, "cell array size: " + std::to_string(cellArraySize));
    log(Priority::Unimportant, "particle array size: " + std::to_string(particleArraySize));
    log(Priority::Unimportant, "auxiliary data size: " + std::to_string(auxiliaryDataSize));

    auto const memorySizeAfter = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();
    log(Priority::Important, std::to_string(memorySizeAfter / (1024 * 1024)) + " MB GPU memory acquired");
}

void _CudaSimulationFacade::checkAndProcessSimulationParameterChanges()
{
    std::lock_guard lock(_mutexForSimulationParameters);
    if (_newSimulationParameters) {
        _settings.simulationParameters = *_newSimulationParameters;
        CHECK_FOR_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &*_newSimulationParameters, sizeof(SimulationParameters), 0, hipMemcpyHostToDevice));
        _newSimulationParameters.reset();

        if (_cudaSimulationData) {
            _simulationKernels->prepareForSimulationParametersChanges(_settings, getSimulationDataIntern());
        }
    }
}

SimulationData _CudaSimulationFacade::getSimulationDataIntern() const
{
    std::lock_guard lock(_mutexForSimulationData);
    return *_cudaSimulationData;
}
