#include "hip/hip_runtime.h"
﻿#include "DebugKernels.cuh"

__device__ void DEBUG_checkCells(SimulationData& data, float* sumEnergy, int location)
{
    auto& cells = data.objects.cellPointers;
    auto partition = calcAllThreadsPartition(cells.getNumEntries());

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        if (auto& cell = cells.at(index)) {

            for (int i = 0; i < cell->numConnections; ++i) {
                auto connectingCell = cell->connections[i].cell;

                auto displacement = connectingCell->absPos - cell->absPos;
                data.cellMap.correctDirection(displacement);
                auto actualDistance = Math::length(displacement);
                if (actualDistance > 14) {
                    printf("distance too large at %d\n", location);
                    CUDA_THROW_NOT_IMPLEMENTED();
                }
            }
            if (cell->energy < 0 || isnan(cell->energy)) {
                printf("cell energy invalid at %d", location);
                CUDA_THROW_NOT_IMPLEMENTED();
            }
            atomicAdd(sumEnergy, cell->energy);
        }
    }
}

__device__ void DEBUG_checkParticles(SimulationData& data, float* sumEnergy, int location)
{
    auto partition = calcPartition(data.objects.particlePointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    for (int particleIndex = partition.startIndex; particleIndex <= partition.endIndex; ++particleIndex) {
        if (auto& particle = data.objects.particlePointers.at(particleIndex)) {
            if (particle->energy < 0 || isnan(particle->energy)) {
                printf("particle energy invalid at %d", location);
                CUDA_THROW_NOT_IMPLEMENTED();
            }
            atomicAdd(sumEnergy, particle->energy);
        }
    }
}

__global__ void DEBUG_checkAngles(SimulationData data)
{
    auto& cells = data.objects.cellPointers;
    auto partition = calcAllThreadsPartition(cells.getNumEntries());

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        if (auto& cell = cells.at(index)) {
            if (cell->numConnections > 0) {
                float sumAngles = 0;
                for (int i = 0; i < cell->numConnections; ++i) {
                    sumAngles += cell->connections[i].angleFromPrevious;
                    if (cell->connections[i].angleFromPrevious < -NEAR_ZERO) {
                        printf("invalid angle: %f\n", cell->connections[i].angleFromPrevious);
                        CUDA_THROW_NOT_IMPLEMENTED();
                    }
                    if (cell->connections[i].angleFromPrevious < NEAR_ZERO) {
                        printf("zero angle\n");
                    }
                }
                if (abs(360.0f - sumAngles) > 0.1f) {
                    printf("invalid angle sum: %f\n", sumAngles);
                    CUDA_THROW_NOT_IMPLEMENTED();
                }
            }
        }
    }
}

__global__ void DEBUG_checkCellsAndParticles(SimulationData data, float* sumEnergy, int location)
{
    DEBUG_checkCells(data, sumEnergy, location);
    DEBUG_checkParticles(data, sumEnergy, location);
}

/*
__global__ void DEBUG_kernel(SimulationData data, int location)
{
    float* sumEnergy = new float;
    *sumEnergy = 0;

    DEPRECATED_KERNEL_CALL_SYNC(DEBUG_checkCellsAndParticles, data, sumEnergy, location);

    float const expectedEnergy = 187500;
    if (abs(*sumEnergy - expectedEnergy) > 1) {
        printf("location: %d, actual energy: %f, expected energy: %f\n", location, *sumEnergy, expectedEnergy);
        CUDA_THROW_NOT_IMPLEMENTED();
    }
    delete sumEnergy;
}
*/
