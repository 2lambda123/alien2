#include "hip/hip_runtime.h"
﻿#include "StatisticsKernels.cuh"

#include "GenomeDecoder.cuh"

__global__ void cudaUpdateTimestepStatistics_substep1(SimulationData data, SimulationStatistics statistics)
{
    statistics.resetTimestepData();
}

__global__ void cudaUpdateTimestepStatistics_substep2(SimulationData data, SimulationStatistics statistics)
{
    {
        auto& cells = data.objects.cellPointers;
        auto const partition = calcAllThreadsPartition(cells.getNumEntries());

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& cell = cells.at(index);
            statistics.incNumCells(cell->color);
            statistics.incNumConnections(cell->color, cell->numConnections);
            statistics.addEnergy(cell->color, cell->energy);
            if (cell->cellFunction == CellFunction_Constructor && GenomeDecoder::containsSelfReplication(cell->cellFunctionData.constructor)) {
                statistics.incNumReplicator(cell->color);
            }
            if (cell->cellFunction == CellFunction_Injector && cell->cellFunctionData.injector.mode == InjectorMode_InjectAll) {
                statistics.incNumReplicator(cell->color);
            }
            if (cell->cellFunction == CellFunction_Constructor) {
                statistics.addNumGenomeBytes(cell->color, cell->cellFunctionData.constructor.genomeSize);
                statistics.incNumGenomes(cell->color);
            }
            if (cell->cellFunction == CellFunction_Injector) {
                statistics.addNumGenomeBytes(cell->color, cell->cellFunctionData.injector.genomeSize);
                statistics.incNumGenomes(cell->color);
            }
        }
    }
    {
        auto& particles = data.objects.particlePointers;
        auto const partition = calcAllThreadsPartition(particles.getNumEntries());

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& particle = particles.at(index);
            statistics.incNumParticles(particle->color);
            statistics.addEnergy(particle->color, particle->energy);
        }
    }
}

__global__ void cudaUpdateTimestepStatistics_substep3(SimulationData data, SimulationStatistics statistics)
{
    statistics.halveNumConnections();
}

__global__ void cudaUpdateHistogramData_substep1(SimulationData data, SimulationStatistics statistics)
{
    statistics.resetHistogramData();
}

__global__ void cudaUpdateHistogramData_substep2(SimulationData data, SimulationStatistics statistics)
{
    auto& cells = data.objects.cellPointers;
    auto const partition = calcAllThreadsPartition(cells.getNumEntries());

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cell = cells.at(index);
        if (cell->barrier) {
            continue;
        }
        statistics.maxValue(cell->age);
    }
}

__global__ void cudaUpdateHistogramData_substep3(SimulationData data, SimulationStatistics statistics)
{
    auto& cells = data.objects.cellPointers;
    auto const partition = calcAllThreadsPartition(cells.getNumEntries());

    auto maxAge = statistics.getMaxValue();
    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cell = cells.at(index);
        if (cell->barrier) {
            continue;
        }
        auto slot = cell->age * MAX_HISTOGRAM_SLOTS / (maxAge + 1);
        statistics.incNumCells(cell->color, slot);
    }
}
